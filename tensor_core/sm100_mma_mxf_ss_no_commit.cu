
#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void sm100_mma_mxf4_ss_test(uint32_t *input, uint32_t *output, uint64_t desc_a,
      uint64_t desc_b,
      uint32_t tmem_c,
      uint32_t scaleC,
      uint64_t idescE,
      uint32_t tsfa_addr,
      uint32_t tsfb_addr) {
    asm volatile(
          "{\n\t"
          ".reg .pred p;\n\t"
          "setp.ne.b32 p, %4, 0;\n\t"
          "tcgen05.mma.cta_group::1.kind::mxf4nvf4.block_scale.scale_vec::4X [%0], %1, %2, %3, [%5], [%6], p; \n\t"
          "}\n"
          :
          : "r"(tmem_c), "l"(desc_a), "l"(desc_b), "r"(uint32_t(idescE>>32)), "r"(scaleC),
            "r"(tsfa_addr), "r"(tsfb_addr));
}

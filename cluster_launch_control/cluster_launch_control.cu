
#include <hip/hip_runtime.h>
#include <cstdint>

struct CLCResponse { uint32_t data[4]; };

__global__ void cluster_launch_control_test(uint32_t* output) {
  __shared__ CLCResponse clc_resp;
  __shared__ uint64_t mbarrier;
  uint32_t result_addr = static_cast<uint32_t>(__cvta_generic_to_shared(&clc_resp));
  uint32_t mbarrier_addr = static_cast<uint32_t>(__cvta_generic_to_shared(&mbarrier));

  asm volatile(
        "{\n\t"
        "clusterlaunchcontrol.try_cancel.async.shared::cta.mbarrier::complete_tx::bytes.multicast::cluster::all.b128 [%0], [%1];\n\t"
        "}\n"
        :
        : "r"(result_addr), "r"(mbarrier_addr));
  uint32_t m_idx, n_idx, l_idx, valid;
  asm volatile(
        "{\n"
        ".reg .pred p1;\n\t"
        ".reg .b128 clc_result;\n\t"
        "ld.shared.b128 clc_result, [%4];\n\t"
        "clusterlaunchcontrol.query_cancel.is_canceled.pred.b128 p1, clc_result;\n\t"
        "selp.u32 %3, 1, 0, p1;\n\t"
        "@p1 clusterlaunchcontrol.query_cancel.get_first_ctaid.v4.b32.b128 {%0, %1, %2, _}, clc_result;\n\t"
        "}\n"
        : "=r"(m_idx), "=r"(n_idx), "=r"(l_idx), "=r"(valid)
        : "r"(result_addr)
        : "memory"
      );
  output[0] = m_idx + n_idx + l_idx + valid;
}



#include <hip/hip_runtime.h>
#include <cstdint>


__global__ void griddep_wait_test(uint32_t* output) {
    asm volatile("griddepcontrol.wait;");
    output[0] = threadIdx.x;
}


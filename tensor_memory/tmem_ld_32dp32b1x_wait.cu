
#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void tmem_ld_test(uint32_t* output, uint32_t tmem_addr) {
  uint32_t output_val;
  asm volatile ("tcgen05.ld.sync.aligned.32x32b.x1.b32"
                    "{%0},"
                    "[%1];\n"
    :  "=r"(output_val)
    :  "r"(tmem_addr));
  
  asm volatile ("tcgen05.wait::ld.sync.aligned;"::);

  output[0] = output_val;
}



#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void tmem_alloc_test(uint32_t* output, uint32_t* alloc_size, int alloc_n) {
  __shared__ uint32_t val;
  for (int i = 0; i < alloc_n; ++i) {
    asm volatile ("tcgen05.alloc.cta_group::1.sync.aligned.shared::cta.b32 [%0], %1;"::"l"(&val), "r"(alloc_size[i]):"memory");
    output[i] = val;
  }
}



#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void tmem_relinquish_test(uint32_t* output, uint32_t* input) {
  uint32_t val = input[0];
  asm volatile ("tcgen05.relinquish_alloc_permit.cta_group::2.sync.aligned;":::"memory");
  output[0] = val;
}


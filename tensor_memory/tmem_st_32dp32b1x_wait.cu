
#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void tmem__test(uint32_t* input, uint32_t* output, uint32_t tmem_addr) {
  uint32_t input_val = input[0];
  asm volatile ("tcgen05.st.sync.aligned.32x32b.x1.b32"
                    "[%0],"
                    "{%1};\n"
    :
    :  "r"(tmem_addr), "r"(input_val) );
  asm volatile (
    "{\n\t"
    "tcgen05.wait::st.sync.aligned; \n"
    "}"
    ::);
  uint32_t output_val;
  asm volatile ("tcgen05.ld.sync.aligned.32x32b.x1.b32"
                    "{%0},"
                    "[%1];\n"
    :  "=r"(output_val)
    :  "r"(tmem_addr));
  output[0] = output_val;
}



#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void tmem_dealloc_test(uint32_t* input) {
  uint32_t val = input[0];
  asm volatile ("tcgen05.dealloc.cta_group::2.sync.aligned.b32 %0, 32;"::"r"(val):"memory");
}


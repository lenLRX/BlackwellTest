
#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void tmem_alloc_test(uint32_t* output) {
  __shared__ uint32_t val;
  asm volatile ("tcgen05.alloc.cta_group::1.sync.aligned.shared::cta.b32 [%0], 64;"::"l"(&val):"memory");
  output[0] = val;
}

